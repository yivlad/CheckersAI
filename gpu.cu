#include "hip/hip_runtime.h"
#define WHITE_KING 'K'
#define BLACK_KING 'Q'
#define CPU_depth 4
#define GPU_depth 3

#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

class Position{
private:
    char board[8][8];
    char move;
    __device__ __host__ void endMove(){
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == 'T') board[i][j] = '#';
            }
        }
        for(int j = 0; j < 8; j++){
            if(board[0][j] == 'W') board[0][j] = WHITE_KING;
            if(board[7][j] == 'B') board[7][j] = BLACK_KING;
        }
        move = move == 'W' ? 'B' : 'W';
    }
    __device__ __host__ void getPossibleJumps(int i, int j, int prevDirection, Position* p, int* n){
        char rival = move == 'W' ? 'B' : 'W';
        char king = move == 'W' ? WHITE_KING : BLACK_KING;
        char rivalking = king == WHITE_KING ? BLACK_KING : WHITE_KING;
        if(board[i][j] == move){
            if(move == 'W' && i == 0) board[i][j] = WHITE_KING;
            if(move == 'B' && i == 7) board[i][j] = BLACK_KING;
            for(int direction = 0; direction < 4; direction++){
                int stepi = ((direction & 1) << 1) - 1,
                    stepj =  (((direction >> 1) & 1) << 1) - 1;
                if(i + 2 * stepi > -1 && j + 2 * stepj > -1 && 
                    i + 2 * stepi < 8 && j + 2 * stepj < 8 &&
                    board[i + 2 * stepi][j + 2 * stepj] == '#'){
                    if(board[i + stepi][j + stepj] == rival || 
                        board[i + stepi][j +stepj] == rivalking){
                        Position branch = *this;
                        branch.board[i][j] = '#';
                        branch.board[i + 2 * stepi][j + 2 * stepj] = move;
                        branch.board[i + stepi][j + stepj] = 'T';
                        int currentN = *n;
                        branch.getPossibleJumps(i + 2 * stepi, j + 2 * stepj, direction, p, n);
                        if(currentN == *n){
                            branch.endMove();
                            p[*n] = branch;
                            (*n)++;
                        }
                    }
                }
            }
        }
        int opposite = 3 - prevDirection;
        if(board[i][j] == king){
            for(int direction = 0; direction < 4; direction++){
                if(direction == opposite) continue;
                int stepi = ((direction & 1) << 1) - 1,
                    stepj =  (((direction >> 1) & 1) << 1) - 1;
                int ri = i + stepi, rj = j + stepj;
                while(ri > -1 && ri < 8 && rj > -1 && rj < 8 && board[ri][rj] == '#')
                {
                    ri += stepi;
                    rj += stepj;
                }
                if(!(ri > -1 && ri < 8 && rj > -1 && rj < 8)) continue;
                if(board[ri][rj] != rival && board[ri][rj] != rivalking){
                    continue;
                }
                int ki = ri + stepi, kj = rj + stepj;
                while(ki > -1 && ki < 8 && kj > -1 && kj < 8 && board[ki][kj] == '#')
                {
                    Position branch = *this;
                    branch.board[i][j] = '#';
                    branch.board[ki][kj] = king;
                    branch.board[ri][rj] = 'T';
                    int currentN = *n;
                    branch.getPossibleJumps(i + 2 * stepi, j + 2 * stepj, direction, p, n);
                    if(currentN == *n){
                        branch.endMove();
                        p[*n] = branch;
                        (*n)++;
                    }
                    ki += stepi;
                    kj += stepj;
                }
            }
        }
    }
    __device__ __host__ bool hasPossibleJumps(int i, int j){
        char rival = move == 'W' ? 'B' : 'W';
        char king = move == 'W' ? WHITE_KING : BLACK_KING;
        char rivalking = king == WHITE_KING ? BLACK_KING : WHITE_KING;
        if(board[i][j] == move){
            if(move == 'W' && i == 0) board[i][j] = WHITE_KING;
            if(move == 'B' && i == 7) board[i][j] = BLACK_KING;
            for(int direction = 0; direction < 4; direction++){
                int stepi = ((direction & 1) << 1) - 1,
                    stepj =  (((direction >> 1) & 1) << 1) - 1;
                if(i + 2 * stepi > -1 && j + 2 * stepj > -1 && 
                    i + 2 * stepi < 8 && j + 2 * stepj < 8 &&
                    board[i + 2 * stepi][j + 2 * stepj] == '#'){
                    if(board[i + stepi][j + stepj] == rival || 
                        board[i + stepi][j +stepj] == rivalking){
                            return true;
                    }
                }
            }
        }
        if(board[i][j] == king){
            for(int direction = 0; direction < 4; direction++){
                int stepi = ((direction & 1) << 1) - 1,
                    stepj =  (((direction >> 1) & 1) << 1) - 1;
                int ri = i + stepi, rj = j + stepj;
                while(ri > -1 && ri < 8 && rj > -1 && rj < 8 && board[ri][rj] == '#')
                {
                    ri += stepi;
                    rj += stepj;
                }
                if(!(ri > -1 && ri < 8 && rj > -1 && rj < 8)) continue;
                if(board[ri][rj] != rival && board[ri][rj] != rivalking){
                    continue;
                }
                int ki = ri + stepi, kj = rj + stepj;
                if(ki > -1 && ki < 8 && kj > -1 && kj < 8 && board[ki][kj] == '#')
                {
                    return true;
                }
            }
        }
        return false;
    }
    __device__ __host__ void getPossibleMoves(Position* p, int* n){
        int direction = move == 'W' ? -1: 1;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == move){
                    if(i + direction < 8 && i + direction > -1){
                        if(j - 1 > 0 && board[i + direction][j - 1] == '#'){
                            Position branch = *this;
                            branch.board[i][j] = '#';
                            branch.board[i + direction][j - 1] = move;
                            branch.endMove();
                            p[*n] = branch;
                            (*n)++;
                        }
                        if(j + 1 < 8 && board[i + direction][j + 1] == '#'){
                            Position branch = *this;
                            branch.board[i][j] = '#';
                            branch.board[i + direction][j + 1] = move;
                            branch.endMove();
                            p[*n] = branch;
                            (*n)++;
                        }
                    }
                }
            }
        }
        char king = move == 'W' ? WHITE_KING : BLACK_KING;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == king){
                    for(direction = 0; direction < 4; direction++){
                        int stepi = ((direction & 1) << 1) - 1,
                            stepj =  (((direction >> 1) & 1) << 1) - 1;
                        int x = i + stepi, y = j + stepj;
                        while(x > -1 && x < 8 && y > -1 && y < 8 && board[x][y] == '#'){
                            Position branch = *this;
                            branch.board[i][j] = '#';
                            branch.board[x][y] = king;
                            branch.endMove();
                            p[*n] = branch;
                            (*n)++;
                            x += stepi;
                            y += stepj;
                        }
                    }
                }
            }
        }
    }
    __device__ __host__ bool hasPossibleMoves(){
        int direction = move == 'W' ? -1: 1;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == move){
                    if(i + direction < 8 && i + direction > -1){
                        if(j - 1 > 0 && board[i + direction][j - 1] == '#'){
                            return true;
                        }
                        if(j + 1 < 8 && board[i + direction][j + 1] == '#'){
                            return true;
                        }
                    }
                }
            }
        }
        char king = move == 'W' ? WHITE_KING : BLACK_KING;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == king){
                    for(direction = 0; direction < 4; direction++){
                        int stepi = ((direction & 1) << 1) - 1,
                            stepj =  (((direction >> 1) & 1) << 1) - 1;
                        int x = i + stepi, y = j + stepj;
                        if(x > -1 && x < 8 && y > -1 && y < 8 && board[x][y] == '#'){
                            return true;
                        }
                    }
                }
            }
        }
        return false;
    }
    __device__ __host__ void getPossibleJumps(Position* p, int* n){
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                getPossibleJumps(i, j, -1, p, n);
            }
        }
    }
public:
    __host__ Position(char board[8][8], char move){
        for(int x = 0; x < 8; x++){
            for(int y = 0; y < 8; y++){
                this->board[x][y] = board[x][y];
            }
        }
        this->move = move;
    }
    __device__ __host__ Position(const Position& another){
        for(int x = 0; x < 8; x++){
            for(int y = 0; y < 8; y++){
                this->board[x][y] = another.board[x][y];
            }
        }
        this->move = another.move;
    }
    __device__ __host__ bool min(){
        if(move == 'B') return true;
        return false;
    }
    __device__ __host__ void getPossiblePositions(Position* p, int* n){
        *n = 0;
        this->getPossibleJumps(p, n);
        if(*n != 0) return;
        this->getPossibleMoves(p, n);
    }
    __device__ __host__ bool hasPossiblePositions(){
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(hasPossibleJumps(i, j)) return true;
            }
        }
        if(hasPossibleMoves()) return true;
        return false;
    }
    __device__ __host__ double evaluate(){
        if(!hasPossiblePositions()){
            if(move == 'W') return -100;
            else return 100;
        }
        bool f = false;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == 'W' || board[i][j] == WHITE_KING) f = true;
            }
        }
        if(!f) return -100;
        f = false;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                if(board[i][j] == 'B' || board[i][j] == BLACK_KING) f = true;
            }
        }
        if(!f) return 100;
        double balance = 0;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                char p = board[i][j];
                if(p == 'W'){
                    double coeff = 1;
                    if(j == 0 || j == 7){
                        coeff *= 0.9;
                    }
                    if(i == 6){
                        coeff *= 1.5;
                    }
                    balance += coeff * 1;
                }
                if(p == 'B'){
                    double coeff = 1;
                    if(j == 0 || j == 7){
                        coeff *= 0.9;
                    }
                    if(i == 1){
                        coeff *= 1.5;
                    }
                    balance += coeff * (-1);
                }
                if(p == WHITE_KING) balance += 3.5;
                if(p == BLACK_KING) balance -= 3.5;
            }
        }
        return balance;
    }
    __host__ friend std::ostream& operator<<(std::ostream& out, const Position& p){
        out << p.move << std::endl;
        for(int i = 0; i < 8; i++){
            for(int j = 0; j < 8; j++){
                out << p.board[i][j];
            }
            out << std::endl;
        }
        return out;
    }
};

class TreeNode {
public:
    Position pos;
    double estimate;
    int index;
    std::vector<TreeNode*> next;
    TreeNode(Position pos) : pos(pos) {
        index = -1;
    }
    void calculatenext(){
        estimate = pos.evaluate();
        if(estimate == 100 || estimate == -100) return;
        Position* p = (Position*) malloc(50 * sizeof(Position));
        int n = 0;
        pos.getPossiblePositions(p, &n);
        for(int i = 0; i < n; i++){
            next.push_back(new TreeNode(p[i]));
        }
        free(p);
    }
};

class Tree{
private:
    int leaves; 
    TreeNode* root;
    void clean(TreeNode* node){
        for(int i =0; i < node->next.size(); i++){
            clean(node->next[i]);
        }
        delete node;
    }
    void expand(TreeNode* node, int depth){
        if(depth == 0){
            node->index = leaves;
            leaves++;
            return;
        }
        node->calculatenext();
        for(int i =0; i < node->next.size(); i++){
            expand(node->next[i], depth - 1);
        }
    }
    void traverse(TreeNode* node, double* estimations){
        if(node->next.size() == 0){
            if(node->index != -1){
                node->estimate = estimations[node->index];
            }
            return;
        }
        double cur;
        if(node->pos.min()) cur = 200;
        else cur = -200;
        for(int i = 0; i < node->next.size(); i++){
            TreeNode* p = node->next[i];
            traverse(p, estimations);
            double e = p->estimate;
            if(node->pos.min()){
                if(e < cur) cur = e;
            }else{
                if(e > cur) cur = e;
            }
        }
        node->estimate = cur;
    }
    void leavestoarray(TreeNode* node, Position* p){
        if(node->index != -1){
            p[node->index] = node->pos;
            return;
        }
        for(int i = 0; i < node->next.size(); i++){
            leavestoarray(node->next[i], p);
        }
    }
public:
    Tree(Position pos, int depth){
        root = new TreeNode(pos);
        leaves = 0;
        expand(root, depth);
    }
    void leavestoarray(Position* p){
        leavestoarray(root, p);
    }
    void traverse(double* estimations){
        traverse(root, estimations);
    }
    int getleaves(){
        return leaves;
    }
    Position bestmove(){
        int index = -1;
        if(root->pos.min()){
            double min = 200;
            for(int i = 0; i < root->next.size(); i++){
                if(root->next[i]->estimate < min){
                    min = root->next[i]->estimate;
                    index = i;
                }
            }
        }
        else
        {
            double max = -200;
            for(int i = 0; i < root->next.size(); i++){
                if(root->next[i]->estimate > max){
                    max = root->next[i]->estimate;
                    index = i;
                }
            }
        }
        return root->next[index]->pos;
    }
    ~Tree(){
        clean(root);
    }
};

class MinimaxStackNode {
public:
    int max, cur;
    double* estimates;
    Position* positions;
    __device__ MinimaxStackNode() {
        positions = (Position*) malloc(50 * sizeof(Position));
        estimates = (double*) malloc(50 * sizeof(double));
    }
    __device__ ~MinimaxStackNode(){
        free(positions);
        free(estimates);
    }
};

__device__ double minimax(Position* pos){
    MinimaxStackNode stack[GPU_depth];
    pos->getPossiblePositions(stack[0].positions, &stack[0].max);
    stack[0].cur = 0;
    int current_node = 0;
    while(current_node >= 0){
        if(stack[current_node].cur < stack[current_node].max){
            int i = stack[current_node].cur;
            double heuristic = stack[current_node].positions[i].evaluate();
            if(current_node < GPU_depth - 1 && heuristic != 100 && heuristic != -100){
                stack[current_node].positions[i].getPossiblePositions(stack[current_node + 1].positions, &stack[current_node + 1].max);
                stack[current_node + 1].cur = 0;
                current_node++;
            }
            else{
                stack[current_node].estimates[i] = heuristic;
                stack[current_node].cur++;
            }
        }else{
            current_node--;
            if(current_node < 0) break;
            if(stack[current_node].positions[stack[current_node].cur].min()){
                double min = 200;
                for(int i = 0; i < stack[current_node + 1].max; i++){
                    if(stack[current_node + 1].estimates[i] < min){
                        min = stack[current_node + 1].estimates[i];
                    }
                }
                stack[current_node].estimates[stack[current_node].cur] = min;
            }
            else{
                double max = -200;
                for(int i = 0; i < stack[current_node + 1].max; i++){
                    if(stack[current_node + 1].estimates[i] > max){
                        max = stack[current_node + 1].estimates[i];
                    }
                }
                stack[current_node].estimates[stack[current_node].cur] = max;
            }
            stack[current_node].cur++;
        }
    }
    if(pos->min()){
        double min = 200;
        for(int i = 0; i < stack[0].max; i++){
            if(stack[0].estimates[i] < min){
                min = stack[0].estimates[i];
            }
        }
        return min;
    }
    else{
        double max = -200;
        for(int i = 0; i < stack[0].max; i++){
            if(stack[0].estimates[i] > max){
                max = stack[0].estimates[i];
            }
        }
        return max;
    }
}

__global__ void checkers_kernel(Position* d_p, double* d_e, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < n){
        d_e[index] = minimax(d_p + index);
    }
}

int main(){
    std::ifstream input;
    input.open("input.txt");
    char board[8][8];
    char move;
    move = input.get();
    input.ignore();
    for(int i = 0; i < 8; i++){
        for(int j = 0; j < 8; j++){
            board[i][j] = input.get();
        }
        input.ignore(1);
    }
    input.close();
    Position pos(board, move);
    double heuristic = pos.evaluate();
    if(heuristic == 100){
        std::cout << "White wins!" << std::endl;
        return EXIT_SUCCESS;
    }
    if(heuristic == -100){
        std::cout << "Black wins!" << std::endl;
        return EXIT_SUCCESS;
    }
    Tree gametree(pos, CPU_depth);
    int l = gametree.getleaves();
    Position* p = (Position*) malloc(sizeof(Position) * l);
    double* h_e;
    h_e = (double*) malloc(sizeof(double) * l);
    gametree.leavestoarray(p);
    Position* d_p;
    double* d_e;
    checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize, 1024 * 4));
    checkCudaErrors(hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 512));
    checkCudaErrors(hipMalloc((void**)&d_p, sizeof(Position) * l));
    checkCudaErrors(hipMalloc((void**)&d_e, sizeof(double) * l));
    checkCudaErrors(hipMemcpy(d_p, p, sizeof(Position) * l, hipMemcpyHostToDevice));
    int threads = 64;
    int blocks = l / threads;
    if(l % threads != 0) blocks++;
    checkers_kernel<<<blocks, threads>>>(d_p, d_e, l);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(h_e, d_e, sizeof(double) * l, hipMemcpyDeviceToHost));
    hipFree(d_e);
    hipFree(d_p);
    gametree.traverse(h_e);
    Position best = gametree.bestmove();
    std::ofstream output;
    output.open("output.txt");
    output << best << std::endl;
    output.close();
    free(p);
    free(h_e);
    return EXIT_SUCCESS;
}